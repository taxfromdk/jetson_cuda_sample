#include "hip/hip_runtime.h"
 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime_api.h>
 #include "cudahelper.h"
 
 // Macro for CUDA error checking - useful for debugging
 #define CUDA_CHECK(call) \
   do { \
     hipError_t err = call; \
     if (err != hipSuccess) { \
       fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
       return 0; \
     } \
   } while(0)
 
 /* CUDA kernel for drawing text on NV12 frame */
 __global__ void drawKernel(
     unsigned char* d_frame,
     int pitch,
     int width,
     int height)
 {
     int x = blockIdx.x * blockDim.x + threadIdx.x;
     int y = blockIdx.y * blockDim.y + threadIdx.y;
     
     if (x >= width || y >= height) return;
     
     if (x < 50 && y < 50) {
         d_frame[y * pitch + x] = 255;
     }
 }
 
 int cuda_process_frame(int gpuId, void* y_plane, uint16_t width, uint16_t height, uint16_t pitch)
 {
    hipError_t err; 
    printf("cuda_process_frame started\n");
     if (y_plane == NULL) {
         fprintf(stderr, "Error: Null CUDA pointer\n");
         return 0;
     }
     
     CUDA_CHECK(hipSetDevice(gpuId));
     
     // Clear any previous errors
     err = hipGetLastError();
     if (err != hipSuccess) {
         printf("CUDA context error before processing: %s\n", hipGetErrorString(err));
     }
     printf("Processing frame with CUDA: width=%d, height=%d, pitch=%d, ptr=%p\n", 
            width, height, pitch, y_plane);
     
    CUDA_CHECK(hipDeviceSynchronize());
     

    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks(
        (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y
    );
    
    printf("Launching kernel with grid: (%d,%d), block: (%d,%d)\n", 
            numBlocks.x, numBlocks.y, threadsPerBlock.x, threadsPerBlock.y);
    
    drawKernel<<<numBlocks, threadsPerBlock>>>(
        (unsigned char*)y_plane,
        pitch,
        width,
        height
    );
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    printf("after kernel\n");
    
     // Wait for GPU to finish
     CUDA_CHECK(hipDeviceSynchronize());
     
     printf("cuda_process_frame completed successfully\n");
     return 1;
 }